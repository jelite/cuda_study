#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//ldm=M(6), n=N(5) p=1 q=2 alpha=16 beta=12
static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    //cublasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    //                 , 3  , &16   , &m[IDX2C(1,2,6)],  6
    //cublasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
    //                 , 6-1=6, 12   , &m[IDX2C(1,2,6)], 1
    hipblasSscal (handle, ldm*n, &beta, &m[IDX2C(0,0,6)], 1);
}

void print_matrix(float *a)
{
    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
}
int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (float)(i * N + j + 1);
        }
    }
    printf("matrix a:\n");
    print_matrix(a);

    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    printf("\nafter matrix a:\n");
    print_matrix(a);
    free(a);
    return EXIT_SUCCESS;
}