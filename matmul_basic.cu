
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define ARR_WIDTH 64
#define TILE_WIDTH 16
#define NUM_BLOCK ARR_WIDTH / TILE_WIDTH

__global__ void matrix_multiply(float* A, float* B, float* C)
{
    int elem;
    for(int i = 0; i < ARR_WIDTH; i++)
    {
        for(int j = 0; j < ARR_WIDTH; j++)
        {
            for(int k = 0; k < ARR_WIDTH; k++)
            {
                elem += A[i*ARR_WIDTH + k]*B[k*ARR_WIDTH + j];
            }
            C[i*ARR_WIDTH + j] = elem;
        }
    }
}

void random_elem(float* arr, int arr_size)
{
  for(int i = 0; i < arr_size; i++)
  {
    for(int j = 0; j < arr_size; j++)
    {
      arr[i*arr_size + j] = (float)rand()/(float(RAND_MAX/10));
    }
  }
}
void print_array(float* arr, int arr_size)
{
  for(int i = 0; i < arr_size; i++)
  {
    for(int j = 0; j < arr_size; j++)
    {
      printf("%f ", arr[i*arr_size + j]);
    }
    printf("\n");
  }
}

int main(void)
{
  float A[ARR_WIDTH*ARR_WIDTH], B[ARR_WIDTH*ARR_WIDTH], C[ARR_WIDTH*ARR_WIDTH];
  float *A_g, *B_g, *C_g;
  random_elem(A, ARR_WIDTH);
  random_elem(B, ARR_WIDTH);

  hipMalloc((float**)&A_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);
  hipMalloc((float**)&B_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);
  hipMalloc((float**)&C_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);

  hipMemcpy(A_g, A, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(B_g, B, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(C_g, C, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);


  // initialize x and y arrays on the kernel
  matrix_multiply<<<1,1>>>(A_g, B_g, C_g);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  hipMemcpy(C, C_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyDeviceToHost);
  // printf("\nmatmul C print:\n");
  return 0;
}