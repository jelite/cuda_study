#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#include<cublas.h>
#include <iostream>

void checkCU(hipblasStatus_t status)
{
    if(status != HIPBLAS_STATUS_SUCCESS) std::cout << "CUDA ERR" << status << std::endl;
    else std::cout << "CUDA SUCCESS" << status << std::endl;
}

int main()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    const int height = 2;
    const int width = 5;
    float *arr = (float*)malloc(sizeof(float)*height*width);
    float *arr_d;
    float alpha = 2;
    if (!arr) {
        std::cout << "host memory allocation failed" << std::endl;
        return EXIT_FAILURE;
    }
    for(int i = 0; i < height*width; i++) arr[i] = i;

    //cublasAlloc(); deprecated; changed to cudaMalloc()
    cudaStat = hipMalloc((void**)&arr_d, height*width*sizeof(float));
    if (cudaStat)
    {
        std::cout << "device memory allocation failed" << std::endl;
    }

    stat = hipblasCreate(&handle);
    if(stat)
    {
        std::cout << "CUBLAS initializtion failed" << std::endl;
    }


    std::cout << cudaStat << std::endl;
    // checkCU(cublasSetMatrix(width, height, sizeof(float), arr, width, arr_d, width)); ////
    // cublasSscal(width*height, alpha, arr_d, 1); ////
    // checkCU(cublasGetMatrix(width, height, sizeof(float), arr_d, width, arr, width)); ////

    // //cudaThreadSynchronize(); deprecated; changed to cudaDeviceSynchronize()
    // cudaDeviceSynchronize();
    // //cublasFree(); deprecated; changed to cudaFree()
    // cudaFree(arr_d);

    // free(arr);

    return 0;
}