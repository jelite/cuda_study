
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define ARR_WIDTH 64
#define TILE_WIDTH 4
#define NUM_BLOCK ARR_WIDTH / TILE_WIDTH

__global__ void tile_matrix_multiply(float* A, float* B, float* C, int width)
{
  //printf("kernel\n");
  __shared__ float shareA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float shareB[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int row = by * blockDim.y + ty;
  int col = bx * blockDim.x + tx;

  float temp = 0;

  //Loop over the M and N tiles required to compute the 'P' element
  for(int i = 0; i < width/TILE_WIDTH; i++){
    shareA[ty][tx] = A[row*width + (i*TILE_WIDTH + tx)];
    shareB[ty][tx] = B[(i*TILE_WIDTH + ty)*width + col];
    __syncthreads();
    for(int k = 0; k < TILE_WIDTH; ++k) temp += shareA[ty][k] * shareB[k][tx];
    __syncthreads(); 
  }
  C[row*width + col] = temp;
}

void random_elem(float* arr, int arr_size)
{
  for(int i = 0; i < arr_size; i++)
  {
    for(int j = 0; j < arr_size; j++)
    {
      arr[i*arr_size + j] = (float)rand()/(float(RAND_MAX/10));
    }
  }
}
void print_array(float* arr, int arr_size)
{
  for(int i = 0; i < arr_size; i++)
  {
    for(int j = 0; j < arr_size; j++)
    {
      printf("%f ", arr[i*arr_size + j]);
    }
    printf("\n");
  }
}

int main(void)
{
  float A[ARR_WIDTH*ARR_WIDTH], B[ARR_WIDTH*ARR_WIDTH], C[ARR_WIDTH*ARR_WIDTH];
  float *A_g, *B_g, *C_g;
  random_elem(A, ARR_WIDTH);
  random_elem(B, ARR_WIDTH);

  hipMalloc((float**)&A_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);
  hipMalloc((float**)&B_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);
  hipMalloc((float**)&C_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH);

  hipMemcpy(A_g, A, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(B_g, B, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(C_g, C, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyHostToDevice);


  // initialize x and y arrays on the kernel
  tile_matrix_multiply<<<dim3(TILE_WIDTH, TILE_WIDTH), dim3(TILE_WIDTH, TILE_WIDTH)>>>(A_g, B_g, C_g, ARR_WIDTH);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  hipMemcpy(C, C_g, sizeof(float) * ARR_WIDTH*ARR_WIDTH, hipMemcpyDeviceToHost);
  // printf("\nmatmul C print:\n");
  return 0;
}